/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/

#include <iostream>
#include <vector>
#include <cmath>
#include <cstdlib>
#include <hip/hip_runtime.h>

static inline void check(hipError_t err, const char *context)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error: " << context << ": "
                  << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

static inline int divup(int a, int b) {
    return (a + b - 1)/b;
}

// static inline int roundup(int a, int b) {
//     return divup(a, b) * b;
// }

#define CHECK(x) check(x, #x)

__global__ void kernel(int ny, int nx, const float *dGPU, float *rGPU)
{
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = threadIdx.y + blockIdx.y * blockDim.y;

    if (row >= ny || col >= ny)
    {
        return;
    }

    if (row <= col)
    {
        // Calculate correlation coefficient
        float sum = 0.0f;
        for (int k = 0; k < nx; k++)
        {
            sum += dGPU[k + row * nx] * dGPU[k + col * nx];
        }
        rGPU[col + row * ny] = sum;
    }
    else
    {
        rGPU[col + row * ny] = 0.0f;
    }
}

void normalize(int ny, int nx, const float *data, std::vector<float> &matrix)
{
    for (int row = 0; row < ny; row++)
    {
        float sum = 0.0f;

        for (int col = 0; col < nx; col++)
        {
            sum += data[col + row * nx];
        }

        float mean = sum / nx;

        float square_sum = 0.0f;

        for (int col = 0; col < nx; col++)
        {
            float x = data[col + row * nx] - mean;
            matrix[col + row * nx] = x;
            square_sum += pow(x, 2);
        }

        square_sum = std::sqrt(square_sum);

        for (int col = 0; col < nx; col++)
        {
            matrix[col + row * nx] /= square_sum;
        }
    }
}

void correlate(int ny, int nx, const float *data, float *result)
{
    std::vector<float> matrix(ny * nx);

    normalize(ny, nx, data, matrix);

    // Allocate memory & copy data to GPU
    float *dGPU = NULL;
    CHECK(hipMalloc((void **)&dGPU, ny * nx * sizeof(float)));
    float *rGPU = NULL;
    CHECK(hipMalloc((void **)&rGPU, ny * ny * sizeof(float)));

    // Transfer data to device
    CHECK(hipMemcpy(dGPU, matrix.data(), ny * nx * sizeof(float), hipMemcpyHostToDevice));

    // Kernel grid
    dim3 dimBlock(16, 16);
    dim3 dimGrid(divup(ny, dimBlock.x), divup(ny, dimBlock.y));

    // Run kernel
    kernel<<<dimGrid, dimBlock>>>(ny, nx, dGPU, rGPU);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    // Copy data back to CPU & release memory
    CHECK(hipMemcpy(result, rGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(rGPU));
}