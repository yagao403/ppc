/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/

#include <iostream>
#include <vector>
#include <cmath>
#include <cstdlib>
#include <hip/hip_runtime.h>

static inline void check(hipError_t err, const char *context)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error: " << context << ": "
                  << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

static inline int divup(int a, int b)
{
    return (a + b - 1) / b;
}

// static inline int roundup(int a, int b) {
//     return divup(a, b) * b;
// }

#define CHECK(x) check(x, #x)

__global__ void kernel(int ny, int nx, const float *dGPU, float *rGPU)
{
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if (col >= ny || row >= ny)
    {
        return;
    }

    if (col < row)
    {
        rGPU[col + row * ny] = 0.0f;
        return;
    }

    // Calculate correlation coefficient
    float sum = 0.0f;
    for (int k = 0; k < nx; k++)
    {
        sum += dGPU[k + col * nx] * dGPU[k + row * nx];
    }
    rGPU[col + row * ny] = sum;
    
}

void normalize(int ny, int nx, const float *data, std::vector<float> &matrix)
{
    for (int row = 0; row < ny; row++) {
        for (int col = 0; col < nx; col++) {
            matrix[col + row * nx] = data[col + row * nx];
        }
    }

    for (int row = 0; row < ny; row++) {
        float rowSum = 0;
        for (int col = 0; col < nx; col++) {
            rowSum += matrix[col + row * nx];
        }
        float rowMean = rowSum / nx;
        for (int col = 0; col < nx; col++) {
            matrix[col + row * nx] -= rowMean;
        }
    }

    for (int row = 0; row < ny; row++) {
        float sumSquare = 0;
        for (int col = 0; col < nx; col++) {
            sumSquare += matrix[col + row * nx] * matrix[col + row * nx];
        }
        float div = std::sqrt(1 / sumSquare);
        for (int col = 0; col < nx; col++) {
            matrix[col + row * nx] = matrix[col + row * nx] * div;
        }
    }
}

void correlate(int ny, int nx, const float *data, float *result)
{
    std::vector<float> matrix(ny * nx);

    normalize(ny, nx, data, matrix);

    // Allocate memory & copy data to GPU
    float *dGPU = NULL;
    CHECK(hipMalloc((void **)&dGPU, ny * nx * sizeof(float)));
    float *rGPU = NULL;
    CHECK(hipMalloc((void **)&rGPU, ny * ny * sizeof(float)));

    // Transfer data to device
    CHECK(hipMemcpy(dGPU, matrix.data(), ny * nx * sizeof(float), hipMemcpyHostToDevice));

    // Kernel grid
    dim3 dimBlock(16, 16);
    dim3 dimGrid(divup(ny, dimBlock.x), divup(ny, dimBlock.y));

    // Run kernel
    kernel<<<dimGrid, dimBlock>>>(ny, nx, dGPU, rGPU);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    // Copy data back to CPU & release memory
    CHECK(hipMemcpy(result, rGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(rGPU));
}